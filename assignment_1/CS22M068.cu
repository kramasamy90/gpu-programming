/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n). 
 * Note: All lines marked in --> should be replaced with code. 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
	long K = blockIdx.x * blockDim.x + threadIdx.x;
	if(K >= m * n) return;
	long int i = K / n;
	long int y = K % n;
	for(long int j = 0; j < n; j++){
		for(long int x = 0; x < m; x++){
			long int I = i * n + y;
			long int J = j * m + x;
			C[I * m * n + J] = A[i * n + j] * B[x * n + y];
		}
	}
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
	long K = blockIdx.x * (blockDim.x * blockDim.y);
	K += threadIdx.x * blockDim.y + threadIdx.y;
	if(K >= m * n) return;
	long int j = K / m;
	long int x = K % m;
	for(long int i = 0; i < m; i++){
		for(long int y = 0; y < n; y++){
			long int I = i * n + y;
			long int J = j * m + x;
			C[I * m * n + J] = A[i * n + j] * B[x * n + y];
		}
	}
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
	long int K = blockIdx.x * gridDim.y + blockIdx.y;
	K *= blockDim.x * blockDim.y;
	K += threadIdx.x * blockDim.y + threadIdx.y;
	if(K >= (n * n * m * m)) return;
	long int I = K / (m * n);
	long int J = K % (m * n);
	long int i = I / n;
	long int j = J / m;
	long int x = J % m;
	long int y = I % i;
	C[K] = A[i * n + j] * B[x * n + y];
}

/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(long int *arr, long int rows, long int cols, char* filename){
    outfile.open(filename);
    for(long int i = 0; i < rows; i++){
        for(long int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    long int m,n;	
    cin>>m>>n;	

    // Host_arrays 
    long int *h_a,*h_b,*h_c;

    // Device arrays 
    long int *d_a,*d_b,*d_c;
	
    // Allocating space for the host_arrays 
    h_a = (long int *) malloc(m * n * sizeof(long int));
    h_b = (long int *) malloc(m * n * sizeof(long int));	
    h_c = (long int *) malloc(m * m * n * n * sizeof(long int));	

    // Allocating memory for the device arrays 
    // --> Allocate memory for A on device 
	hipMalloc(&d_a, m * n * sizeof(long int));
    // --> Allocate memory for B on device 
	hipMalloc(&d_b, m * n * sizeof(long int));
    // --> Allocate memory for C on device 
	hipMalloc(&d_c, m * m * n * n * sizeof(long int));

    // Read the input matrix A 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_a[i];
    }

    //Read the input matrix B 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_b[i];
    }

    // Transfer the input host arrays to the device 
    // --> Copy A from Host to Device
	hipMemcpy(d_a, h_a, m*n*sizeof(long int), hipMemcpyHostToDevice);
    // --> Copy B from Host to Device 
	hipMemcpy(d_b, h_b, m*n*sizeof(long int), hipMemcpyHostToDevice);

    long int gridDimx, gridDimy;
    
    // Launch the kernels
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    // --> Set the launch configuration 
	gridDimx = ceil((float(m * n)/ 1024));
	gridDimy = 1;
	dim3 grid_1(gridDimx, gridDimy, 1);
	dim3 block_1(1024, 1, 1);

    double starttime = rtclock();  

    // --> Launch the kernel 
	per_row_AB_kernel<<<grid_1, block_1>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize();                                                           

    double endtime = rtclock(); 
	printtime("GPU Kernel-1 time: ", starttime, endtime);  

    // --> Copy C from Device to Host 
	hipMemcpy(h_c, d_c, (m * m * n * n)*sizeof(long int), 
		hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/
    
    // --> Set the launch configuration 
	gridDimx = ceil((float(m * n)/ 1024));
	gridDimy = 1;
	dim3 grid_2(gridDimx, gridDimy, 1);
	dim3 block_2(64, 16, 1);

    starttime = rtclock(); 

    // --> Launch the kernel 
	per_column_AB_kernel<<<grid_2, block_2>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize(); 

    endtime = rtclock(); 
  	printtime("GPU Kernel-2 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
	hipMemcpy(h_c, d_c, (m * m * n * n)*sizeof(long int), 
		hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output 
     **/
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid3(gridDimx,gridDimy,1);
    dim3 block3(64,16,1);

    starttime = rtclock();  

    // --> Launch the kernel 
	per_element_kernel<<<grid3, block3>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize();                                                              

    endtime = rtclock();  
	printtime("GPU Kernel-3 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
	hipMemcpy(h_c, d_c, (m * m * n * n)*sizeof(long int), 
		hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel3.txt");

    return 0;
}